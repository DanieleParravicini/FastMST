#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <vector>
#include <iostream>
#include "Graph.h"
#include "CompactGraph.h"
#include <thrust\fill.h>
#include <thrust\partition.h>
#include <thrust\scan.h>


int main()
{
	std::vector<std::vector<int>> weights = { {0,2},{2, 0} };
	Graph g(weights, weights.size());
	std::cout << g.to_string();
	std::cout << g.toCompact().to_string();
	try {
		mst(g);
	}
	catch (...) {
		std::cout << "Some error occurred :'(";
	}

    return 0;
}

void mst(Graph g) {
	CompactGraph c = g.toCompact();

	mst(c);
}

struct compactGraphOnGpu {
	int* vertices = 0;
	int* edgePtr = 0;
	int* weights = 0;
	int* edges = 0;
	int numEdges;
	int numVertices;
	int* X;
};
void mst(CompactGraph g) {
	const int blockFactor = 32;

	compactGraphOnGpu onGPU;
	onGPU.numEdges = g.edges.size();
	onGPU.numVertices = g.vertices.size();

	hipError_t status;
	try {
		//1. obtain space where to put the data structure that represent the graph.
		status = hipMalloc(&onGPU.vertices, sizeof(int)*g.vertices.size());
		if (status != hipError_t::hipSuccess)
			throw status;
		status = hipMalloc(&onGPU.edgePtr, sizeof(int)*g.edgePtr.size());
		if (status != hipError_t::hipSuccess)
			throw status;
		status = hipMalloc(&onGPU.weights, sizeof(int)*g.weights.size());
		if (status != hipError_t::hipSuccess)
			throw status;
		status = hipMalloc(&onGPU.edges, sizeof(int)*g.edges.size());
		if (status != hipError_t::hipSuccess)
			throw status;
		//2. transfer data to GPU memory
		status = hipMemcpy(onGPU.vertices, &g.vertices[0], sizeof(int)*g.vertices.size(), hipMemcpyKind::hipMemcpyHostToDevice);
		if (status != hipError_t::hipSuccess)
			throw status;
		status = hipMemcpy(onGPU.edgePtr, &g.edgePtr[0], sizeof(int)*g.edgePtr.size(), hipMemcpyKind::hipMemcpyHostToDevice);
		if (status != hipError_t::hipSuccess)
			throw status;
		status = hipMemcpy(onGPU.weights, &g.weights[0], sizeof(int)*g.weights.size(), hipMemcpyKind::hipMemcpyHostToDevice);
		if (status != hipError_t::hipSuccess)
			throw status;
		status = hipMemcpy(onGPU.edges, &g.edges[0], sizeof(int)*g.edges.size(), hipMemcpyKind::hipMemcpyHostToDevice);
		if (status != hipError_t::hipSuccess)
			throw status;
		//3. once memory transfer has been achievede we have to find 
		//   for each vertex the min cost outgoing edge
		minOutgoingEdge(onGPU);
		

		//4. rebuild compact graph representation for next algorithm iteration.


	}
	catch (hipError_t err) {
		std::cout << err;
		hipFree(onGPU.vertices);
		hipFree(onGPU.edgePtr);
		hipFree(onGPU.weights);
		hipFree(onGPU.edges);

		throw err;
	}


	hipFree(onGPU.vertices);
	hipFree(onGPU.edgePtr);
	hipFree(onGPU.weights);
	hipFree(onGPU.edges);
	
}

int* copy() {
	/*
	status = hipMalloc(&onGPU.vertices, sizeof(int)*g.vertices.size());
	if (status != hipError_t::hipSuccess)
		throw status;
	status = hipMemcpy(onGPU.vertices, &g.vertices[0], sizeof(int)*g.vertices.size(), hipMemcpyKind::hipMemcpyHostToDevice);
	if (status != hipError_t::hipSuccess)
		throw status;
		*/
}
int* MarkEdgeSegments(compactGraphOnGpu onGPU) {
	hipError_t status;
	int * flags;
	try {
		status = hipMalloc(&flags, sizeof(int)*onGPU.numEdges);
		if (status != hipError_t::hipSuccess)
			throw status;
		fill << <onGPU.numEdges / 32, 32 >> >(flags, 0, onGPU.numEdges);
		mark_edge_ptr << <onGPU.numVertices / 32, 32 >> >(flags, onGPU.edgePtr, onGPU.numVertices);
		
	}
	catch (...) {
		hipFree(flags);
		throw status;
	}
	return flags;
}

void segmentedScan(int* out, int* in, int * flags, int width) {
	int * tmpKeys;
	hipMalloc(&tmpKeys, sizeof(int)*width);
	hipMemcpy(tmpKeys, flags, width, hipMemcpyKind::hipMemcpyHostToDevice);
	thrust::inclusive_scan(tmpKeys, tmpKeys + width, tmpKeys);
	thrust::inclusive_scan_by_key(tmpKeys, tmpKeys + width, in, out);
}

struct min {
	__host__ __device__
	int  operator()(const int a, const int b) const 
	{ 
		return a < b ? a: b; 
	}
};

void segmentedMinScan(int* out, int* in, int* flags, int width) {
	int * tmpKeys;
	hipMalloc(&tmpKeys, sizeof(int)*width);
	hipMemcpy(tmpKeys, flags, width, hipMemcpyKind::hipMemcpyHostToDevice);
	thrust::inclusive_scan(tmpKeys, tmpKeys + width, tmpKeys);

	thrust::equal_to<int> binary_pred;
	min binary_op;

	thrust::inclusive_scan_by_key(tmpKeys, tmpKeys + width, in, out, binary_pred, binary_op);
}

void split(int* data, int* flags, int width) {
	int * tmpKeys;
	hipMalloc(&tmpKeys, sizeof(int)*width);
	hipMemcpy(tmpKeys, flags, width, hipMemcpyKind::hipMemcpyHostToDevice);
	thrust::inclusive_scan(tmpKeys, tmpKeys + width, tmpKeys);
	thrust::sort_by_key(tmpKeys, tmpKeys + width, data);
}

__global__ void fill(int* out, int immediate, int width) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < width) {
		out[idx] = immediate;
	}
}
__global__ void mark_edge_ptr(int* out, int* ptr, int width) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < width) {
		out[ptr[idx]] = 1;
	}
}
__global__ void getMinNodes() {

}