#include "hip/hip_runtime.h"
#include "kernel.h"


#define BLOCK_SIZE 16
#define DEBUG
#define GRID(N, B) ((N + B - 1) / B); 

int grid(int n, int block) {
	return (n + block - 1) / block;
}

int main()
{
	std::vector<std::vector<int>> weights = { {0, 1, 2} ,{2, 0, 0} ,{0, 0, 0} };
	Graph g(weights, weights.size());
	std::cout << g.to_string();
	std::cout << g.toCompact().to_string();
	try {
		mst(g);
	}
	catch (...) {
		std::cout << "Some error occurred :'(";
	}

    return 0;
}


void mst(Graph g) {
	CompactGraph c = g.toCompact();
	mst(c);
}

void mst(CompactGraph g) {

	DatastructuresOnGpu onGPU;
	onGPU.numEdges = g.edges.size();
	onGPU.numVertices = g.vertices.size();

	try {
		//1. move data structures to GPU memory
		onGPU.vertices = (int*)moveToGpu(g.vertices);
		onGPU.edgePtr = (int*)moveToGpu(g.edgePtr);
		onGPU.edges = (int*)moveToGpu(g.edges);
		onGPU.weights = (int*)moveToGpu(g.weights);

		mst(onGPU);
	}
	catch (hipError_t err) {
		std::cout << err;
		hipFree(onGPU.vertices);
		hipFree(onGPU.edgePtr);
		hipFree(onGPU.weights);
		hipFree(onGPU.edges);

		throw err;
	}


	hipFree(onGPU.vertices);
	hipFree(onGPU.edgePtr);
	hipFree(onGPU.weights);
	hipFree(onGPU.edges);

}

void mst(DatastructuresOnGpu onGPU) {
	if (onGPU.numVertices == 1)
		return;

	//1. for each vertex we have to find 
	//   the min cost outgoing edge
	minOutgoingEdge(onGPU);


	//3. rebuild compact graph representation for next algorithm iteration.

	//4. update vertices count. by using scan result.
	//4. recall mst();
}


/// on X in position pointed by edgePtr[src] we can obtain the min couple (weight, dest) for that src vertex.
void minOutgoingEdge(DatastructuresOnGpu onGPU) {
	hipError_t status;
	status = hipMalloc(&onGPU.X, sizeof(int)*onGPU.numEdges);
	if (status != hipError_t::hipSuccess)
		throw status;

	fill << < grid(onGPU.numEdges , BLOCK_SIZE), BLOCK_SIZE >> >(onGPU.X, onGPU.edges, onGPU.numEdges, createMask(0, 22));
	#ifdef DEBUG
		std::cout << "X after fill with destination vertices from bit 0 to 22: " + debug_device_ptr(onGPU.X, onGPU.numEdges);
	#endif
	
	fill << < grid(onGPU.numEdges, BLOCK_SIZE), BLOCK_SIZE >> >(onGPU.X, onGPU.weights, onGPU.numEdges, createMask(22, 10), 22);
	#ifdef DEBUG
		std::cout << "X after fill with weights from bit 22 to 32: " + debug_device_ptr(onGPU.X, onGPU.numEdges);
	#endif
	status = hipMalloc(&onGPU.F, sizeof(int)*onGPU.numEdges);
	if (status != hipError_t::hipSuccess)
		throw status;
	fill << <grid(onGPU.numEdges, BLOCK_SIZE), BLOCK_SIZE >> >(onGPU.F, 0, onGPU.numEdges);
	mark_edge_ptr << < grid(onGPU.numEdges, BLOCK_SIZE), BLOCK_SIZE >> >(onGPU.F, onGPU.edgePtr, onGPU.numVertices);
	#ifdef DEBUG
	{
		std::cout << "F before segmented min scan: " + debug_device_ptr(onGPU.F, onGPU.numEdges);
	}
	#endif
	segmentedMinScanInCuda(onGPU.X, onGPU.X, onGPU.F, onGPU.numEdges);
	#ifdef DEBUG
	{
		std::cout << "F after segmented min scan: " + debug_device_ptr(onGPU.F, onGPU.numEdges);
		std::cout << "X after segmented min scan: "+ debug_device_ptr(onGPU.X, onGPU.numEdges);
	}
	#endif
}



int* MarkEdgeSegments(DatastructuresOnGpu onGPU) {
	hipError_t status;
	int * flags;
	try {
		status = hipMalloc(&flags, sizeof(int)*onGPU.numEdges);
		if (status != hipError_t::hipSuccess)
			throw status;
		MarkEdgeSegmentsOnGpu(onGPU, flags);

	}
	catch (...) {
		hipFree(flags);
		throw status;
	}
	return flags;
}

void MarkEdgeSegmentsOnGpu(DatastructuresOnGpu onGPU, int* flags) {
	fill<< < grid(onGPU.numEdges, BLOCK_SIZE), BLOCK_SIZE >> >(flags, 0, onGPU.numEdges);
	mark_edge_ptr<<< grid(onGPU.numVertices, BLOCK_SIZE), BLOCK_SIZE >> >(flags, onGPU.edgePtr, onGPU.numVertices);
}