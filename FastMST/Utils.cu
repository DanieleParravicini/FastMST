#include "hip/hip_runtime.h"
#include "Utils.h"



int grid(int n, int block) {
	return (n + block - 1) / block;
}

__global__ void copyIndirected(unsigned int* dst, unsigned int* src, unsigned int * ptr, unsigned int n, unsigned int m) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if (idx < n - 1) {
		dst[idx] = src[ptr[idx + 1] - 1];
	}
	else if (idx = n - 1) {
		dst[idx] = src[m - 1];
	}
}


template<int mask, int shift>
int unMaskAndShift(int a);


template< int op(int)>
__global__ void copy(int* src, int* dst, int n) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if (idx < n) {
		dst[idx] = op(src[idx]);
	}
}

__global__ void moveWeightsAndSuccessors(unsigned int* src, unsigned int* dstW, unsigned int * dstS, unsigned int n) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if (idx < n) {
		int successor = src[idx] & createMask(0, VERTEX_SIZE);
		int successor_of_successor = src[successor] & createMask(0, VERTEX_SIZE);

		int weight;
		if (idx < successor && idx == successor_of_successor) {
			weight = 0;
			successor = idx;
		}
		else {
			weight = (src[idx] & createMask(VERTEX_SIZE, WEIGHT_SIZE)) >> VERTEX_SIZE;
		}
		dstW[idx] = weight;
		dstS[idx] = successor;
	}
}


unsigned int* MarkEdgeSegments(DatastructuresOnGpu onGPU) {
	hipError_t status;
	unsigned int * flags;
	try {
		status = hipMalloc(&flags, sizeof(unsigned int)*onGPU.numEdges);
		if (status != hipError_t::hipSuccess)
			throw status;
		MarkEdgeSegmentsOnGpu(onGPU, flags);

	}
	catch (...) {
		hipFree(flags);
		throw status;
	}
	//error
	return flags;
}

void MarkEdgeSegmentsOnGpu(DatastructuresOnGpu onGPU, unsigned int* flags) {
	hipMemset(flags, 0, onGPU.numEdges* sizeof(unsigned int));
	mark_edge_ptr << <  grid(onGPU.numVertices, BLOCK_SIZE), BLOCK_SIZE >> > (flags, onGPU.edgePtr, onGPU.numVertices);
}


__host__ __device__ unsigned int createMask(int start, int width) {
	unsigned int mask = 1;
	mask = (mask << width) - 1;
	mask = mask << start;
	return mask;
}

void* moveToGpu(void* src, size_t sizeinBytes) {
	void* dst;
	hipError_t	status;
	try {
		status = hipMalloc(&dst, sizeinBytes);
		if (status != hipError_t::hipSuccess)
			throw status;
		status = hipMemcpy(dst, src, sizeinBytes, hipMemcpyKind::hipMemcpyHostToDevice);
		if (status != hipError_t::hipSuccess)
			throw status;
	}
	catch (...) {
		hipFree(dst);
		throw status;
	}
	return dst;
}

void* moveToGpu(unsigned int* src, unsigned int size) {
	return moveToGpu((void*)src, sizeof(unsigned int)*size);
}

void* moveToGpu(std::vector<unsigned int> src) {
	return moveToGpu((void*)&src[0], sizeof(unsigned int)*src.size());
}

void debug_device_ptr(unsigned int* ptr, unsigned  int items) {

	unsigned int* x;
	x = (unsigned int*)malloc(sizeof(unsigned int)*items);
	hipMemcpy(x, ptr, sizeof(unsigned int)*items, hipMemcpyKind::hipMemcpyDeviceToHost);

	for (unsigned int i = 0; i < items; i++) {
		std::cout << std::to_string(x[i]) << ";";
	}
	std::cout << "\n";
	free(x);
}

void debug_device_ptr(unsigned int* ptr, unsigned int items, unsigned int nrbit) {

	unsigned int* x;
	x = (unsigned int*)malloc(sizeof(unsigned int)*items);
	hipMemcpy(x, ptr, sizeof(unsigned int)*items, hipMemcpyKind::hipMemcpyDeviceToHost);

	for (unsigned int i = 0; i < items; i++) {
		unsigned int mask = createMask(0, nrbit);
		std::cout << "<" << std::to_string((x[i] & ~mask) >> nrbit) << "," << std::to_string(x[i] & mask) << ">" << ";";
	}
	std::cout << "\n";
	free(x);
}


struct min_op {
	__host__ __device__
		int  operator()(const int a, const int b) const
	{
		return (a < b) ? a : b;
	}
};

void segmentedMinScan(unsigned int* out, unsigned int* in, unsigned int* flags, unsigned int width) {
	//this function is only a stub for testing purposes
	unsigned int * tmpKeys;
	hipMalloc(&tmpKeys, sizeof(unsigned int)*width);
	hipMemcpy(tmpKeys, flags, width, hipMemcpyKind::hipMemcpyHostToDevice);

	segmentedMinScanInCuda(out, in, tmpKeys, width);

	hipFree(tmpKeys);
}

///
/// flags will be touched and invalidated. flags will contain vertex identifier
void segmentedMinScanInCuda(unsigned int* out, unsigned int* in, unsigned int* flags, unsigned int width) {

	thrust::device_ptr<unsigned int> dev_in(in);
	thrust::device_ptr<unsigned int> dev_out(out);
	thrust::device_ptr<unsigned int> dev_flags(flags);

	thrust::inclusive_scan(dev_flags, dev_flags + width, dev_flags);

	thrust::equal_to<unsigned int> binary_pred;
	thrust::minimum <unsigned int> binary_op;

	thrust::inclusive_scan_by_key(dev_flags, dev_flags + width, dev_in, dev_out, binary_pred, binary_op);
}

template<typename Tdst, typename Tsrc>
__global__ void fill(Tdst* out, Tsrc* src, int width, int from) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < width) {
		out[idx] = (src[idx] << from) | (out[idx]);
	}
}

__global__ void fill(unsigned int* out, unsigned int immediate, unsigned int width) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < width) {
		out[idx] = immediate;
	}
}

__global__ void fill(unsigned int* out, unsigned int* src, unsigned int width, unsigned int mask) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < width) {
		out[idx] = (src[idx] & mask) | (out[idx] & ~mask);
	}
}

__global__ void fill(unsigned int* out, unsigned int* src, unsigned int width, unsigned int mask, unsigned int from) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < width) {
		out[idx] = ((src[idx] << from) & mask) | (out[idx] & ~mask);
	}
}

__global__ void mark_edge_ptr(unsigned int* out, unsigned int* ptr, unsigned int width) {
	//first idx start at 0, which should not be marked!
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx > 0 && idx < width) {
		out[ptr[idx]] = 1;
	}
}

__global__ void mark_discontinuance(unsigned int* out, unsigned int* ptr, unsigned int width) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx + 1 < width) {
		if (ptr[idx] != ptr[idx + 1]) {
			out[idx + 1] = 1;
		}
	}
}


__global__ void replaceTillFixedPointInShared(unsigned int * S, unsigned int n) {
	//Move S of block into a shared memory.
	extern __shared__ unsigned int A[] ;
	int pos;
	int items_per_thread = (n + blockDim.x - 1) / blockDim.x;	
#ifdef PEDANTIC
	if (threadIdx.x == 0) {
		printf("total : %d elements; %d items for each thread", n, items_per_thread);
	}
#endif
	for (int i = 0; i < items_per_thread ; i++) {
		pos = threadIdx.x * items_per_thread + i;
		if (pos < n) {
			A[pos] = S[pos];
		}
	}

	__shared__ bool flag;
	unsigned int new_s;
	unsigned int old_s;
	do {
		__syncthreads();
		if (threadIdx.x == 0) {
			flag = false;
		}

		for (int i = 0; i < items_per_thread ; i++) {
			pos = threadIdx.x*items_per_thread + i;
			if (pos < n) {
				old_s = A[pos];
				new_s = A[old_s];


				if (old_s != new_s) {
					flag = true;
					A[pos] = new_s;
				}
			}
			__syncthreads();
		}
		
	} while (flag);


	for (int i = 0; i < items_per_thread ; i++) {
		pos = (threadIdx.x * items_per_thread)  + i;
		//printf("[thread : %d of %d ] pos %d \n ", threadIdx.x, blockDim.x, pos);
		if (pos < n) {
			S[pos] = A[pos];
		}
	}
	

}

__device__ unsigned long long toCacheCell(unsigned int key, unsigned int value) {
	return (key << 16) << 16 || value;
}

__device__ unsigned int keyCacheCell(unsigned long long cell) {
	unsigned int ret = (cell >> 16) >> 16;
	return ret;
}

__device__ unsigned int valueCacheCell(unsigned long long cell) {
	unsigned int ret = cell & createMask(0, 32);
	return ret;
}

__global__ void replaceTillFixedPoint(unsigned int * S, unsigned int n) {
	//the shared memory is used as a cache
	//first 32 bit store the key, last 32 bit store a value.


	int pos, pos_in_cache;
	int items_per_thread = (n + blockDim.x - 1) / blockDim.x;
	

#ifdef PEDANTIC
	if (threadIdx.x == 0) {
		printf("total : %d elements; %d items for each thread", n, items_per_thread);
	}
#endif
	
	bool flag;
	unsigned int new_s;
	unsigned int old_s;
	for (int i = 0; i < items_per_thread; i++) {
		pos = threadIdx.x*items_per_thread + i;
	
		if (threadIdx.x == 0) {
			flag = false;
		}


		old_s = S[pos];
		do {
				
				new_s = S[old_s];
				
				if (old_s != new_s) {
					flag = true;
					//S[pos] = new_s;
				}
				old_s = new_s;
			
		} while (pos < n && old_s != new_s);

	}

}

__global__ void mark_differentUV(unsigned int* flag, unsigned int* v, unsigned int* e, unsigned int n) {
	//TODO: consider go with vertex indeces [1, n+1].
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx == 0) {
		//in case of zero it's enough not to be a self loop
		if (v[idx] != e[idx])
			flag[0] = 1;
	}
	else if (idx < n) {
		//we have to pick edges that are not self-loops 
		//and which have different (source, destination)
		if (v[idx] != e[idx] &&
			(v[idx - 1] != v[idx] || e[idx - 1] != e[idx])) {
			flag[idx] = 1;
		}

	}

}

__global__ void mark_differentU(unsigned int* flag, unsigned int* v, unsigned int n) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < n -1) {
		if (v[idx] != v[idx + 1]) {
			flag[idx] = 1;
		}

	}

}

struct minus1 : public thrust::unary_function<unsigned int, unsigned int>
{
	__host__ __device__
		unsigned int operator()(unsigned int x)
	{

		return x - 1;
	}
};
struct plus1 : public thrust::unary_function<unsigned int, unsigned int>
{
	__host__ __device__
		unsigned int operator()(unsigned int x)
	{
		
		return x + 1;
	}
};

/// on X in position pointed by edgePtr[src] we can obtain the min couple (weight, dest) for that src vertex.
void minOutgoingEdge(DatastructuresOnGpu* onGPU) {

	fill << < grid(onGPU->numEdges, BLOCK_SIZE), BLOCK_SIZE >> >(onGPU->X, onGPU->edges	 , 
		onGPU->numEdges, createMask(0, VERTEX_SIZE));
	//or hipMemcpy(onGPU->X, onGPU->edges, sizeof(unsigned int)* onGPU->numEdges, hipMemcpyDeviceToDevice);
	hipDeviceSynchronize();
	fill << < grid(onGPU->numEdges, BLOCK_SIZE), BLOCK_SIZE >> >(onGPU->X, onGPU->weights,
		onGPU->numEdges, createMask(VERTEX_SIZE, WEIGHT_SIZE), VERTEX_SIZE);
	
	#ifdef PEDANTIC
	std::cout << "X after fill with weights from bit "<< VERTEX_SIZE <<" to " << VERTEX_SIZE + WEIGHT_SIZE <<"; and edge ids from bit 0 to "<< VERTEX_SIZE << std::endl;
	debug_device_ptr(onGPU->X, onGPU->numEdges, VERTEX_SIZE);
	#endif

	hipMemset(onGPU->F, 0, sizeof(unsigned int) * onGPU->numEdges);
	hipDeviceSynchronize();
	mark_edge_ptr << < grid(onGPU->numEdges, BLOCK_SIZE), BLOCK_SIZE >> >(onGPU->F, onGPU->edgePtr, onGPU->numVertices);
	hipDeviceSynchronize();
	segmentedMinScanInCuda(onGPU->X, onGPU->X, onGPU->F, onGPU->numEdges);
	hipDeviceSynchronize();
	#ifdef PEDANTIC
	std::cout << "Source vertex index in vertex list: " << std::endl;
	debug_device_ptr(onGPU->F, onGPU->numEdges);
	std::cout << "X after segmented min scan: " << std::endl;
	debug_device_ptr(onGPU->X, onGPU->numEdges, VERTEX_SIZE);
	#endif
}


void moveMinWeightsAndSuccessor(DatastructuresOnGpu* onGPU) {
	//we move to S: the couples (W,V) with minimum W. 

	copyIndirected <<< grid(onGPU->numVertices, BLOCK_SIZE), BLOCK_SIZE >>>(onGPU->F, onGPU->X, onGPU->edgePtr, onGPU->numVertices, onGPU->numEdges);
	hipDeviceSynchronize();

	#ifdef PEDANTIC
	std::cout << "====================== Move min weights and successor ====================" << std::endl;
	std::cout << "X after segmented min scan:  (w,v)" << std::endl;
	debug_device_ptr(onGPU->F, onGPU->numEdges, VERTEX_SIZE);
	std::cout << "EdgePtr " << std::endl;
	debug_device_ptr(onGPU->edgePtr, onGPU->numVertices);
	#endif
	#if defined(PEDANTIC) || defined(DEBUG) 
	std::cout << "Min outgoing edge for each node (in F) (w,v)" << std::endl;
	debug_device_ptr(onGPU->F, onGPU->numVertices, VERTEX_SIZE);
	#endif
	//eliminate cycles in s[s[i]] = i;
	//unpack weights and outgoing edge in X, S.
	// at the same time perform eliminate cycles S[S[i]] = i
	//move successors and weight in S and X respectively.
	// S[S[i]] = i are addressed
	moveWeightsAndSuccessors << < grid(onGPU->numVertices, BLOCK_SIZE), BLOCK_SIZE >> >(onGPU->F, onGPU->X, onGPU->S, onGPU->numVertices);
	hipDeviceSynchronize();
}



void computeCosts(DatastructuresOnGpu* onGPU) {
	#ifdef PEDANTIC
		std::cout << "weights of min cut:" << std::endl;
		debug_device_ptr(onGPU->X, onGPU->numVertices);
	#endif
	//use weights stored in X to
	//compute additional cost of this step of min spanning tree.
	//note: moveWeights set to 0 elements S[S[i]] = i;

	thrust::device_ptr<unsigned int> Ws(onGPU->X);
	thrust::inclusive_scan(Ws, Ws + onGPU->numVertices, Ws);
	hipDeviceSynchronize();

#ifdef PEDANTIC
	std::cout << "sum of min weights outgoing edges performed on X (w,v)" << std::endl;
	debug_device_ptr(onGPU->X, onGPU->numVertices);
#endif
	//last cell of F contains the cost.
	//Load into a variable.
	unsigned int deltaCosts = 0;
	hipMemcpy(&deltaCosts, onGPU->X + onGPU->numVertices - 1, sizeof(unsigned int), hipMemcpyDeviceToHost);
#if defined(DEBUG ) || defined(PEDANTIC)
	std::cout << "Additional costs" << deltaCosts << std::endl;
#endif
	//update the current MST cost.
	onGPU->cost += deltaCosts;
}

void buildSuccessor(DatastructuresOnGpu* onGPU) {
#ifdef PEDANTIC
	std::cout << "Successor: " << std::endl;
	debug_device_ptr(onGPU->S, onGPU->numVertices);
#endif
	//compute successor
	if (sizeof(unsigned int) * onGPU->numVertices < 48000) {
		int block_dim = std::min(1024, ((int)(onGPU->numVertices + 16) / 17));
		replaceTillFixedPointInShared << <1, block_dim, sizeof(unsigned int) * onGPU->numVertices >> >(onGPU->S, onGPU->numVertices);
	}
	else {
		replaceTillFixedPoint << <1, 1024, 48000 >> >(onGPU->S, onGPU->numVertices);
	}
	hipDeviceSynchronize();

#ifdef PEDANTIC
	std::cout << "Successor after fixed point: " << std::endl;
	debug_device_ptr(onGPU->S, onGPU->numVertices);
#endif
}

void buildSupervertexId(DatastructuresOnGpu* onGPU) {
	thrust::device_ptr<unsigned int>
		s_on_gpu(onGPU->S), v_on_gpu(onGPU->vertices),
		f_on_gpu(onGPU->F);
	//1. aggregate vertices with the same supervertex id
	//   in order to be able to reconstruct which supervertex is related to which vertex
	//	 we move according to the transformation the original position in which is was. 
	//	 The original position will be temporarely stored in v.
#ifdef PEDANTIC
	std::cout << "Rebuild graph representation for the next iteration" << std::endl;
	std::cout << "Source: " << std::endl;
	debug_device_ptr(onGPU->vertices, onGPU->numVertices);
	std::cout << "Renaming to supervertex: " << std::endl;
	debug_device_ptr(onGPU->S, onGPU->numVertices);
#endif
	fill <<< grid(onGPU->numVertices, BLOCK_SIZE), BLOCK_SIZE >>>(onGPU->vertices, 1, onGPU->numVertices);
	hipDeviceSynchronize();
	//use exclusive scan so that first element get 0 as original position
	thrust::exclusive_scan(v_on_gpu, v_on_gpu + onGPU->numVertices, v_on_gpu);
	hipDeviceSynchronize();
	//2. order according to successor and move accordingly the array v
	thrust::sort_by_key(s_on_gpu, s_on_gpu + onGPU->numVertices, v_on_gpu);
	//3. create a flag that contains 1 in position i , when  supervertex[i-1] != supervertex[i] , 0 otherwise.
	hipMemset(onGPU->F, 0, onGPU->numVertices * sizeof(unsigned int));
	hipDeviceSynchronize();
	mark_discontinuance <<< grid(onGPU->numVertices, BLOCK_SIZE), BLOCK_SIZE >>>(onGPU->F, onGPU->S, onGPU->numVertices);
	hipDeviceSynchronize();

#ifdef PEDANTIC
	std::cout << "Pick different supervertex" << std::endl;
	std::cout << "vertex index: " << std::endl;
	debug_device_ptr(onGPU->vertices, onGPU->numVertices);
	std::cout << "supervertex: " << std::endl;
	debug_device_ptr(onGPU->S, onGPU->numVertices);
	std::cout << "mark supervetex end: " << std::endl;
	debug_device_ptr(onGPU->F, onGPU->numVertices);
#endif

	//4. perform a scan to build supervertex identifier.
	thrust::inclusive_scan(f_on_gpu, f_on_gpu + onGPU->numVertices, f_on_gpu);
	hipDeviceSynchronize();

#ifdef PEDANTIC
	std::cout << "id of supervetex: " << std::endl;
	debug_device_ptr(onGPU->F, onGPU->numVertices);
#endif
	//5. once that for each vertex we have determined the new supervetex id.
	//	 Reconstruct a map that permits to move from the vertex->new_supervertex_id. 
	//   The map will be placed in S. 
	//use v that have been reordered to recreate a map v->new_supervertex_id according to the original order.

	thrust::scatter(f_on_gpu, f_on_gpu + onGPU->numVertices, v_on_gpu, s_on_gpu);
	hipDeviceSynchronize();
#ifdef PEDANTIC
	std::cout << "vertex ->id of supervetex: " << std::endl;
	debug_device_ptr(onGPU->S, onGPU->numVertices);
#endif
}

void orderUVW(DatastructuresOnGpu* onGPU) {
	thrust::device_ptr<unsigned int>
		s_on_gpu(onGPU->S), v_on_gpu(onGPU->vertices),
		f_on_gpu(onGPU->F), e_on_gpu(onGPU->edges),
		w_on_gpu(onGPU->weights), eptr_on_gpu(onGPU->edgePtr),
		x_on_gpu(onGPU->X);

	//1. consider the arrays that describe the edges u-w->v.
	//1.a remap u,v with new supervertex id.
	//	v = source
	//	edges = destination
	//	weights = weights
	hipMemset(onGPU->F, 0, onGPU->numEdges* sizeof(unsigned  int));
	//since elements 0 has to obtain flag = 0 use edgePTR excluding the first cell in the mark_edge_ptr call
	mark_edge_ptr << <grid(onGPU->numVertices, BLOCK_SIZE), BLOCK_SIZE >> > (onGPU->F, onGPU->edgePtr, onGPU->numVertices);
	hipDeviceSynchronize();
#ifdef PEDANTIC
	std::cout << "=============================== reorder according to U,V,W where U-w->V : ===============" << std::endl;
	std::cout << "Edge ptr: " << std::endl;
	debug_device_ptr(onGPU->edgePtr, onGPU->numVertices);
	std::cout << "set 1 at start of the edge segments to build source vertex: " << std::endl;
	debug_device_ptr(onGPU->F, onGPU->numEdges);
#endif
	//after inclusive scan f[i] = index of source vertex
	
	thrust::inclusive_scan(f_on_gpu, f_on_gpu + onGPU->numEdges, f_on_gpu);
	hipDeviceSynchronize();
#ifdef PEDANTIC
	std::cout << "source of the edge:" << std::endl;
	debug_device_ptr(onGPU->F, onGPU->numEdges);
	std::cout << "destination of the edge:" << std::endl;
	debug_device_ptr(onGPU->edges, onGPU->numEdges);
	std::cout << "Map:" << std::endl;
	debug_device_ptr(onGPU->S, onGPU->numVertices);
#endif
	//convert v thanks to vector S.
	thrust::gather(f_on_gpu, f_on_gpu + onGPU->numEdges, s_on_gpu, v_on_gpu);
	thrust::gather(e_on_gpu, e_on_gpu + onGPU->numEdges, s_on_gpu, x_on_gpu);
	//we would like to store all info in e.
	//mem cpy is necessary due to the fact that gather input /output has to reside on different memory areas
	hipMemcpy(onGPU->edges, onGPU->X, sizeof(unsigned int)*onGPU->numEdges, hipMemcpyDeviceToDevice);

#ifdef PEDANTIC
	std::cout << "Transformated source of the edge (Result)  " << std::endl;
	debug_device_ptr(onGPU->vertices, onGPU->numEdges);
	std::cout << "Transformated destination of the edge (Result)" << std::endl;
	debug_device_ptr(onGPU->edges, onGPU->numEdges);
#endif
	//2. reored triplet u,v,w. But instead of using a single array . 
	//	 Exploit a stable sort and a perform three different order carrying along a index represented by X.
	//   as in radix sort go from least signicants bits to most significative bits,
	//2.a. create index increment.
	fill << < grid(onGPU->numEdges, BLOCK_SIZE), BLOCK_SIZE >> >(onGPU->X, 1, onGPU->numEdges);
	hipDeviceSynchronize();
	//2.b. create index by accumulating increments: use exclusive since first element has index 0.
	thrust::exclusive_scan(x_on_gpu, x_on_gpu + onGPU->numEdges, x_on_gpu);
	hipDeviceSynchronize();
#ifdef PEDANTIC
	std::cout << "Keys(X) before ordering" << std::endl;
	debug_device_ptr(onGPU->X, onGPU->numEdges);
	std::cout << "Source vertices" << std::endl;
	debug_device_ptr(onGPU->vertices, onGPU->numEdges);
	std::cout << "Destination vertices" << std::endl;
	debug_device_ptr(onGPU->edges, onGPU->numEdges);
	std::cout << "Weights" << std::endl;
	debug_device_ptr(onGPU->weights, onGPU->numEdges);
#endif
	//3.c. stable_sort w.r.t. weights and carry along the modification on indices
	//NOTE: exploit S as a temporary buffer in order to not corrupt the original array. this dose not hold for the last order.
	hipMemcpy(onGPU->S, onGPU->weights, sizeof(unsigned int)*onGPU->numEdges, hipMemcpyDeviceToDevice);
	hipDeviceSynchronize();
	thrust::stable_sort_by_key(s_on_gpu, s_on_gpu + onGPU->numEdges, x_on_gpu);
	hipDeviceSynchronize();
#ifdef PEDANTIC
	std::cout << "Keys(X) after ordering w.r.t weights" << std::endl;
	debug_device_ptr(onGPU->X, onGPU->numEdges);
	std::cout << "weights" << std::endl;
	debug_device_ptr(onGPU->S, onGPU->numEdges);
#endif
	//3.c. before ordering w.r.t. destination vertices first apply reordering according to the index.
	thrust::gather(x_on_gpu, x_on_gpu + onGPU->numEdges, e_on_gpu, s_on_gpu);
	hipDeviceSynchronize();
#ifdef PEDANTIC
	std::cout << "Keys(X) before ordering w.r.t destination vertices" << std::endl;
	debug_device_ptr(onGPU->X, onGPU->numEdges);
	std::cout << "destination vertex" << std::endl;
	debug_device_ptr(onGPU->S, onGPU->numEdges);
#endif
	thrust::stable_sort_by_key(s_on_gpu, s_on_gpu + onGPU->numEdges, x_on_gpu);
	hipDeviceSynchronize();
#ifdef PEDANTIC
	std::cout << "Keys(X) after ordering w.r.t destination vertices" << std::endl;
	debug_device_ptr(onGPU->X, onGPU->numEdges);
	std::cout << "destination vertex" << std::endl;
	debug_device_ptr(onGPU->S, onGPU->numEdges);
#endif
	//3.d. once again before ordering w.r.t. source vertices first apply reordering according to the index.
	//here is not necessary to exploit a temporary buffer since no other reoreder is needed.

	thrust::gather(x_on_gpu, x_on_gpu + onGPU->numEdges, v_on_gpu, s_on_gpu);
	hipDeviceSynchronize();
#ifdef PEDANTIC
	std::cout << "Keys(X) after ordering w.r.t source vertices" << std::endl;
	debug_device_ptr(onGPU->X, onGPU->numEdges);
	std::cout << "source vertex" << std::endl;
	debug_device_ptr(onGPU->S, onGPU->numEdges);
#endif
	thrust::stable_sort_by_key(s_on_gpu, s_on_gpu + onGPU->numEdges, x_on_gpu);
	hipDeviceSynchronize();
	hipMemcpy(onGPU->vertices, onGPU->S, sizeof(unsigned int)*onGPU->numEdges, hipMemcpyDeviceToDevice);
	hipDeviceSynchronize();
#ifdef PEDANTIC
	std::cout << "Keys(X) after ordering w.r.t source vertices" << std::endl;
	debug_device_ptr(onGPU->X, onGPU->numEdges);
	std::cout << "source vertex" << std::endl;
	debug_device_ptr(onGPU->vertices, onGPU->numEdges);
#endif
	//3.e. last: in order for i that:	f[i] < f[j] and 
	//								edges[i] < edges[j] and
	//								weights[i] < weights[j] 
	// we have to reorder edges and weights arrays according to index x.
	//NOTE: nothing has been said about gather values and output possible overlapping.
	hipMemcpy(onGPU->S, onGPU->edges, sizeof(unsigned int)*onGPU->numEdges, hipMemcpyDeviceToDevice);
	hipDeviceSynchronize();
	thrust::gather(x_on_gpu, x_on_gpu + onGPU->numEdges, s_on_gpu, e_on_gpu);
	hipDeviceSynchronize();
	hipMemcpy(onGPU->S, onGPU->weights, sizeof(unsigned int)*onGPU->numEdges, hipMemcpyDeviceToDevice);
	hipDeviceSynchronize();
	thrust::gather(x_on_gpu, x_on_gpu + onGPU->numEdges, s_on_gpu, w_on_gpu);
	hipDeviceSynchronize();

#ifdef PEDANTIC
	std::cout << "Keys(X) after ordering source, destination, weights" << std::endl;
	debug_device_ptr(onGPU->X, onGPU->numEdges);
	std::cout << "Source vertices" << std::endl;
	debug_device_ptr(onGPU->vertices, onGPU->numEdges);
	std::cout << "Destination vertices" << std::endl;
	debug_device_ptr(onGPU->edges, onGPU->numEdges);
	std::cout << "Weights" << std::endl;
	debug_device_ptr(onGPU->weights, onGPU->numEdges);
#endif
}

void rebuildEdgeWeights(DatastructuresOnGpu* onGPU) {
	thrust::device_ptr<unsigned int>
		s_on_gpu(onGPU->S), v_on_gpu(onGPU->vertices),
		f_on_gpu(onGPU->F), e_on_gpu(onGPU->edges),
		w_on_gpu(onGPU->weights), eptr_on_gpu(onGPU->edgePtr),
		x_on_gpu(onGPU->X);


	//8. create Edge, weights
	//8.a use F to mark subseqent (v,u) which are not equals and neither self-loops.
	hipMemset(onGPU->F, 0, onGPU->numEdges* sizeof(unsigned int));
	hipDeviceSynchronize();
	mark_differentUV << <grid(onGPU->numEdges, BLOCK_SIZE), BLOCK_SIZE >> >(onGPU->F, onGPU->vertices, onGPU->edges, onGPU->numEdges);
	hipDeviceSynchronize();
	//8.b perform a scan of F to obtain position were to put edge[idx], weight[idx] put the scan result on X
	//	  then use: 
	//		-	x as a map 
	//		-   F as a stencil
	thrust::inclusive_scan(f_on_gpu, f_on_gpu + onGPU->numEdges, x_on_gpu);
	//8.c compute the number of edges available in the next iteration by reading the tail of the scan vector result.

	hipMemcpy(&onGPU->newNumEdges, onGPU->X + onGPU->numEdges - 1, sizeof(unsigned int), hipMemcpyDeviceToHost); 
	//8.d. since index start at 0 subtract 1 from all the indices.
	thrust::transform(x_on_gpu, x_on_gpu + onGPU->numEdges, x_on_gpu, minus1());
	hipDeviceSynchronize();
#ifdef PEDANTIC
	std::cout << "new edges/weights" << std::endl;
	std::cout << "mark relevant edges for edges/weights" << std::endl;
	debug_device_ptr(onGPU->F, onGPU->numEdges);
	std::cout << "positions" << std::endl;
	debug_device_ptr(onGPU->X, onGPU->numEdges);
#endif
	//Since there are multiple elements with the same destination index and we have to insert only those that are in correspondence with a change in x.
	//we exploit a version of scatter that expect a stencil.
	//8.d. move weights
	//since input data and output can't reside on the same memory area we move them temporarily to S
	hipMemcpy(onGPU->S, onGPU->weights, sizeof(unsigned int)* onGPU->numEdges, hipMemcpyDeviceToDevice);
	hipDeviceSynchronize();
	thrust::scatter_if(s_on_gpu, s_on_gpu + onGPU->numEdges, x_on_gpu, f_on_gpu, w_on_gpu);
	//8.e. move destination vertices
	//since input data and output can't reside on the same memory area we mve them temporarily to S
	hipMemcpy(onGPU->S, onGPU->edges, sizeof(unsigned int)* onGPU->numEdges, hipMemcpyDeviceToDevice);
	hipDeviceSynchronize();
	thrust::scatter_if(s_on_gpu, s_on_gpu + onGPU->numEdges, x_on_gpu, f_on_gpu, e_on_gpu);
	hipDeviceSynchronize();

#ifdef PEDANTIC
	std::cout << "new edges (" << onGPU->newNumEdges << ") :" << std::endl;
	std::cout << "new weights" << std::endl;
	debug_device_ptr(onGPU->weights, onGPU->newNumEdges);
	std::cout << "new edges" << std::endl;
	debug_device_ptr(onGPU->edges, onGPU->newNumEdges);
#endif
}

void rebuildEdgePtr(DatastructuresOnGpu* onGPU) {
	thrust::device_ptr<unsigned int>
		s_on_gpu(onGPU->S), v_on_gpu(onGPU->vertices),
		f_on_gpu(onGPU->F),
		w_on_gpu(onGPU->weights), eptr_on_gpu(onGPU->edgePtr),
		x_on_gpu(onGPU->X);
	//exploit data already computed for Edges in X number of useful edges
	//9. create EdgePTR:
	//9.a find discontinuance in source vertex
	//    since there was a collapsing in the previous step there would always be a edge for each supervertex.
	hipMemset(onGPU->F, 0, onGPU->numEdges* sizeof(unsigned int));
	hipDeviceSynchronize();
	mark_differentU << <grid(onGPU->numEdges, BLOCK_SIZE), BLOCK_SIZE >> >(onGPU->F, onGPU->vertices, onGPU->numEdges);
	hipDeviceSynchronize();
	//9.b perform a inclusive scan on such a flag in order to obtain offset in the edge ptr
	thrust::inclusive_scan(f_on_gpu, f_on_gpu + onGPU->numEdges, s_on_gpu);
	//9.c obtain vertex number by reading from the tail of S
	//and then summing 1
	hipMemcpy(&onGPU->newNumVertices, onGPU->S + onGPU->numEdges - 1, sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	onGPU->newNumVertices++;
	
	//9.d Build edge ptr:
	//when the source vector change pick the number of relevant edges till the previous vertex and then increment by one 
	//9.d.1
	//if on x we have the counted the number of edges that will be preserved
	//   on f we have the stencil that indicate two different consequence sources
	//   on s we have the scan result on the stencil.
#ifdef PEDANTIC
	std::cout << "mark relevant edges for vertices" << std::endl;
	debug_device_ptr(onGPU->F, onGPU->numEdges);
	std::cout << "positions" << std::endl;
	debug_device_ptr(onGPU->S, onGPU->numEdges);
	std::cout << "scan result on relevant edges (already performed at step 7)" << std::endl;
	debug_device_ptr(onGPU->X, onGPU->numEdges);
#endif

	thrust::scatter_if(x_on_gpu, x_on_gpu + onGPU->numEdges, s_on_gpu, f_on_gpu, eptr_on_gpu);
	hipDeviceSynchronize();
	//9.d.2.
	//sum 1 to all to pick up the next
	thrust::transform(eptr_on_gpu, eptr_on_gpu + onGPU->newNumVertices, eptr_on_gpu, plus1());
	hipDeviceSynchronize();
	//set edgeptr of 0 equal to 0.
	hipMemset(onGPU->edgePtr, 0, sizeof(unsigned int));
	hipDeviceSynchronize();
#ifdef PEDANTIC
	std::cout << "new edge ptr" << std::endl;
	debug_device_ptr(onGPU->edgePtr, onGPU->newNumVertices);
#endif



}

void rebuildVertices(DatastructuresOnGpu* onGPU) {
	thrust::device_ptr<unsigned int>
		v_on_gpu(onGPU->vertices),
		x_on_gpu(onGPU->X);

	
	fill<<< grid(onGPU->newNumVertices, BLOCK_SIZE), BLOCK_SIZE>>>(onGPU->X, 1, onGPU->newNumVertices);
	hipDeviceSynchronize();

	thrust::exclusive_scan(x_on_gpu, x_on_gpu + onGPU->newNumVertices, v_on_gpu);
	hipDeviceSynchronize();

#ifdef PEDANTIC
	std::cout << "new vertices (" << onGPU->newNumVertices << ") :" << std::endl;
	debug_device_ptr(onGPU->vertices, onGPU->newNumVertices);
#endif



}






