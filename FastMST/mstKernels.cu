#include "hip/hip_runtime.h"
#include "mstKernels.cuh"

__global__ void moveWeightsAndSuccessors(unsigned int* src, unsigned int* dstW, unsigned int * dstS, unsigned int n) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if (idx < n) {
		int successor = src[idx] & createMask(0, VERTEX_SIZE);
		int successor_of_successor = src[successor] & createMask(0, VERTEX_SIZE);

		int weight;
		if (idx < successor && idx == successor_of_successor) {
			weight = 0;
			successor = idx;
		}
		else {
			weight = (src[idx] & createMask(VERTEX_SIZE, WEIGHT_SIZE)) >> VERTEX_SIZE;
		}
		dstW[idx] = weight;
		dstS[idx] = successor;
	}
}


void MarkEdgeSegmentsOnGpu(DatastructuresOnGpu onGPU, unsigned int* flags) {
	hipMemset(flags, 0, onGPU.numEdges* sizeof(unsigned int));
	mark_edge_ptr <<<  grid(onGPU.numVertices, BLOCK_SIZE), BLOCK_SIZE >> > (flags, onGPU.edgePtr, onGPU.numVertices);
}


__global__ void mark_edge_ptr(unsigned int* out, unsigned int* ptr, unsigned int width) {
	//first idx start at 0, which should not be marked!
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx > 0 && idx < width) {
		out[ptr[idx]] = 1;
	}
}

__global__ void mark_discontinuance(unsigned int* out, unsigned int* ptr, unsigned int width) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx + 1 < width) {
		if (ptr[idx] != ptr[idx + 1]) {
			out[idx + 1] = 1;
		}
	}
}


__global__ void replaceTillFixedPointInShared(unsigned int * S, unsigned int n) {
	//Move S of block into a shared memory.
	extern __shared__ unsigned int A[];
	int pos;
	int items_per_thread = (n + blockDim.x - 1) / blockDim.x;
#ifdef PEDANTIC
	if (threadIdx.x == 0) {
		printf("total : %d elements; %d items for each thread", n, items_per_thread);
	}
#endif
	for (int i = 0; i < items_per_thread; i++) {
		pos = threadIdx.x * items_per_thread + i;
		if (pos < n) {
			A[pos] = S[pos];
		}
	}

	__shared__ bool flag;
	unsigned int new_s;
	unsigned int old_s;
	do {
		__syncthreads();
		if (threadIdx.x == 0) {
			flag = false;
		}

		for (int i = 0; i < items_per_thread; i++) {
			pos = threadIdx.x*items_per_thread + i;
			if (pos < n) {
				old_s = A[pos];
				new_s = A[old_s];


				if (old_s != new_s) {
					flag = true;
					A[pos] = new_s;
				}
			}
			__syncthreads();
		}

	} while (flag);


	for (int i = 0; i < items_per_thread; i++) {
		pos = (threadIdx.x * items_per_thread) + i;
		//printf("[thread : %d of %d ] pos %d \n ", threadIdx.x, blockDim.x, pos);
		if (pos < n) {
			S[pos] = A[pos];
		}
	}


}

__global__ void replaceTillFixedPoint(unsigned int * S, unsigned int n , unsigned int maxShared) {
	//the shared memory is used as a cache
	//first maxShared items are stored in shared memory instead of global memory
	int pos;
	int items_per_thread = (n + blockDim.x - 1) / blockDim.x;
	extern __shared__ unsigned int cache[];

#ifdef PEDANTIC
	if (threadIdx.x == 0) {
		printf("total : %d elements; %d items for each thread", n, items_per_thread);
	}
#endif

	for (int i = 0; i < items_per_thread; i++) {
		pos = threadIdx.x * items_per_thread + i;
		if (pos < maxShared) {
			cache[pos] = S[pos];
		}
	}

	__syncthreads();

	unsigned int new_s;
	unsigned int old_s;
	for (int i = 0; i < items_per_thread; i++) {
		pos = threadIdx.x*items_per_thread + i;
		
		if (pos < n) {

			if (pos < maxShared) {
				new_s = cache[pos];
			}
			else {
				new_s = S[pos];
			}
			
			do {

				old_s = new_s;
				if (old_s < maxShared) {
					new_s = cache[old_s];
				}
				else {
					new_s = S[old_s];
				}

			} while (old_s != new_s);

			
		}
		__syncthreads();

		if (pos < n ) {
			S[pos] = new_s;
		}

		__syncthreads();
	}

}

__global__ void loadNVE(NVEcell * NVE, unsigned int* v, unsigned int * e, unsigned int *w, unsigned int n) {
	
	int pos;
	int items_per_thread = (n + blockDim.x - 1) / blockDim.x;

	for (int i = 0; i < items_per_thread; i++) {
		pos = threadIdx.x*items_per_thread + i;
		if (pos < n) {
			NVE[pos].setSource(v[pos]);
		}
	}
	for (int i = 0; i < items_per_thread; i++) {
		pos = threadIdx.x*items_per_thread + i;
		if (pos < n) {
			NVE[pos].setDestination(e[pos]);
		}
	}
	for (int i = 0; i < items_per_thread; i++) {
		pos = threadIdx.x*items_per_thread + i;
		if (pos < n) {
			NVE[pos].setWeight(w[pos]);
		}
	}

}
__global__ void loadNVEsingle(NVEcell * NVE, unsigned int* v, unsigned int * e, unsigned int *w, unsigned int n) {

	int pos = blockDim.x * blockIdx.x + threadIdx.x;
	
	if (pos < n) {
		NVE[pos].setSource(v[pos]);
		NVE[pos].setDestination(e[pos]);
		NVE[pos].setWeight(w[pos]);
	}


}

__global__ void unloadNVE(NVEcell * NVE, unsigned int* v, unsigned int * e, unsigned int *w, unsigned int n) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < n) {
		v[idx] = NVE[idx].getSource();
		e[idx] = NVE[idx].getDestination();
		w[idx] = NVE[idx].getWeight();
		//printf("v %d e %d w %d : result %d", v[idx], e[idx], w[idx], NVE[idx].cell);
	}
}


__global__ void mark_differentUV(unsigned int* flag, unsigned int* v, unsigned int* e, unsigned int n) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx == 0) {
		//in case of zero it's enough not to be a self loop
		if (v[idx] != e[idx])
			flag[0] = 1;
	}
	else if (idx < n) {
		//we have to pick edges that are not self-loops 
		//and which have different (source, destination)
		if (v[idx] != e[idx] &&
			(v[idx - 1] != v[idx] || e[idx - 1] != e[idx])) {
			flag[idx] = 1;
		}

	}

}

__global__ void mark_differentU(unsigned int* flag, unsigned int* v, unsigned int n) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if ((idx == 0) || (idx < n - 1  && v[idx] != v[idx + 1])) {
		flag[idx+1] = 1;
	}

}

struct minus1 : public thrust::unary_function<unsigned int, unsigned int>
{
	__host__ __device__
		unsigned int operator()(unsigned int x)
	{

		return x - 1;
	}
};

struct plus1 : public thrust::unary_function<unsigned int, unsigned int>
{
	__host__ __device__
		unsigned int operator()(unsigned int x)
	{

		return x + 1;
	}
};

__global__ void loadUV(UVcell* UV, unsigned int* e, unsigned int* w, unsigned int* id, unsigned int n) {
	int pos;
	int items_per_thread = (n + blockDim.x - 1) / blockDim.x;

	for (int i = 0; i < items_per_thread ; i++) {
		pos = threadIdx.x*items_per_thread + i;
		if (pos < n) {
			UV[pos].setDestination(e[pos]);
		}
	}

	for (int i = 0; i < items_per_thread; i++) {
		pos = threadIdx.x*items_per_thread + i;
		if (pos < n) {
			UV[pos].setWeight(w[pos]);
		}
	}

	for (int i = 0; i < items_per_thread; i++) {
		pos = threadIdx.x*items_per_thread + i;
		if (pos < n) {
			UV[pos].setID(id[pos]);
		}
	}
}

__global__ void loadUVsingle(UVcell* UV, unsigned int* e, unsigned int* w, unsigned int* id, unsigned int n) {
	int pos = blockDim.x* blockIdx.x + threadIdx.x;


	if (pos < n) {
		UV[pos].setDestination(e[pos]);
		
		UV[pos].setWeight(w[pos]);
		
		UV[pos].setID(id[pos]);
	}
	
}

struct minimum_UV_cell : public thrust::minimum<UVcell>
{
	__host__ __device__ UVcell operator()(const UVcell &lhs, const UVcell &rhs) {
		
		return lhs.UW < rhs.UW ? lhs : rhs;
	}

};

int blockRounder(int block) {
	return ((block + BLOCK_SIZE - 1 ) / BLOCK_SIZE)*BLOCK_SIZE;
}

/// on X in position pointed by edgePtr[src] we can obtain the min couple (weight, dest) for that src vertex.
void minOutgoingEdge(DatastructuresOnGpu* onGPU) {
	thrust::device_ptr<unsigned int > v_on_gpu(onGPU->vertices);
	thrust::device_ptr<UVcell> uv_on_gpu((UVcell*)onGPU->NVE);

	hipMemset(onGPU->NVE, 0, sizeof(NVEcell)* onGPU->numEdges);

	int block_dim = std::min(1024, blockRounder((int)(onGPU->numVertices + 16) / 17));

	loadUVsingle<<< grid(onGPU->numEdges, BLOCK_SIZE), BLOCK_SIZE >>>((UVcell *) onGPU->NVE, onGPU->edges, onGPU->weights, onGPU->edgeID, onGPU->numEdges);

	hipDeviceSynchronize(); 

	thrust::equal_to<unsigned int> binary_pred;
	minimum_UV_cell binary_op;

	thrust::inclusive_scan_by_key(v_on_gpu, v_on_gpu + onGPU->numEdges, uv_on_gpu, uv_on_gpu, binary_pred, binary_op);
	
	hipDeviceSynchronize(); 
	
}

__global__ void unloadUV(UVcell* UV, unsigned int* dstV, unsigned int* dstID, unsigned int * dstW, unsigned int* edgePtr, unsigned int numvertices, unsigned int numedges) {
	int pos;
	int items_per_thread  = (numvertices + blockDim.x*gridDim.x - 1) / blockDim.x*gridDim.x;


	for (int i = 0; i < items_per_thread; i++) {
		pos = ( blockIdx.x+ threadIdx.x) * items_per_thread + i;
		if (pos < numvertices -1 ) {
			//printf("element : %d value %d \n", pos, edgePtr[pos + 1] - 1);

			dstV[pos] = UV[edgePtr[pos +1 ] - 1].getDestination();
		}
		else if (pos == numvertices -1 ) {
			dstV[pos] = UV[numedges -1].getDestination();
		}

	}

	for (int i = 0; i < items_per_thread; i++) {
		pos = (blockIdx.x + threadIdx.x) * items_per_thread + i;
		if (pos < numvertices - 1) {
			dstID[pos] = UV[edgePtr[pos + 1] -1 ].getID();
		}
		else if (pos == numvertices -1 ) {
			dstID[pos] = UV[numedges-1 ].getID();
		}
	}

	for (int i = 0; i < items_per_thread; i++) {
		pos = (blockIdx.x + threadIdx.x) * items_per_thread + i;
		if (pos < numvertices - 1) {
			dstW[pos] = UV[edgePtr[pos + 1] - 1].getWeight();
		}
		else if (pos == numvertices -1 ) {
			dstW[pos] = UV[numedges-1].getWeight();
		}
	}
	
}

__global__ void unloadUVsingle(UVcell* UV, unsigned int* dstV, unsigned int* dstID, unsigned int * dstW, unsigned int* edgePtr, unsigned int numvertices, unsigned int numedges) {
	int pos = blockDim.x * blockIdx.x + threadIdx.x;

		if (pos < numvertices - 1) {
			//printf("element : %d value %d \n", pos, edgePtr[pos + 1] - 1);

			dstV[pos] = UV[edgePtr[pos + 1] - 1].getDestination();
		}
		else if (pos == numvertices - 1) {
			dstV[pos] = UV[numedges - 1].getDestination();
		}

		if (pos < numvertices - 1) {
			dstID[pos] = UV[edgePtr[pos + 1] - 1].getID();
		}
		else if (pos == numvertices - 1) {
			dstID[pos] = UV[numedges - 1].getID();
		}

		if (pos < numvertices - 1) {
			dstW[pos] = UV[edgePtr[pos + 1] - 1].getWeight();
		}
		else if (pos == numvertices - 1) {
			dstW[pos] = UV[numedges - 1].getWeight();
		}
	

}

__global__ void eliminateSelfLoops(UVcell* UV, unsigned int* dstV, unsigned int* dstID, unsigned int * dstW, unsigned int* edgePtr, unsigned int numvertices, unsigned int numedges) {
	int pos;
	int items_per_thread = (numvertices + blockDim.x*gridDim.x - 1) / blockDim.x*gridDim.x;

	for (int i = 0; i < items_per_thread; i++) {
		pos = (blockIdx.x + threadIdx.x) * items_per_thread + i;
		if (pos < numvertices) {

			int successor = dstV[pos];
			int successor_of_successor = dstV[successor];

			int weight = dstW[pos];
			int id = dstID[pos];

			if (pos < successor && pos == successor_of_successor) {
				successor = pos;
				weight = 0;
				id = 1 << 31;
			}

			dstW[pos] = weight;
			dstV[pos] = successor;
			dstID[pos] = id;
		}
	}
}

__global__ void eliminateSelfLoopSingle(UVcell* UV, unsigned int* dstV, unsigned int* dstID, unsigned int * dstW, unsigned int* edgePtr, unsigned int numvertices, unsigned int numedges) {
	int pos = blockDim.x * blockIdx.x + threadIdx.x;
	if (pos < numvertices) {

		int successor = dstV[pos];
		int successor_of_successor = dstV[successor];

		int weight = dstW[pos];
		int id = dstID[pos];

		if (pos < successor && pos == successor_of_successor) {
			successor = pos;
			weight = 0;
			id = 1 << 31;
		}

		dstW[pos] = weight;
		dstV[pos] = successor;
		dstID[pos] = id;
	}
	
}

void moveMinWeightsAndSuccessor(DatastructuresOnGpu* onGPU) {
	//we move to F the ids of the selected min outgoing edges
	//on X we move the weight
	//on S the destination of the min outgoing edge. 
	int gridDim = grid(onGPU->numVertices, BLOCK_SIZE);
	unloadUVsingle <<< gridDim, BLOCK_SIZE >>>((UVcell *)onGPU->NVE, onGPU->S, onGPU->F ,onGPU->X, onGPU->edgePtr, onGPU->numVertices, onGPU->numEdges);

	hipDeviceSynchronize(); 

	//we have to eliminate cycles in s[s[i]] = i;
	//unpack weights and outgoing edge in X, S respectively .
	eliminateSelfLoopSingle <<< gridDim, BLOCK_SIZE >> >((UVcell *)onGPU->NVE, onGPU->S, onGPU->F, onGPU->X, onGPU->edgePtr, onGPU->numVertices, onGPU->numEdges);
	#if defined(PEDANTIC) || defined(DEBUG) 

	#endif
	
}


void computeCosts(DatastructuresOnGpu* onGPU) {
#ifdef PEDANTIC
	std::cout << "weights of min outgoing edges for each vertex:" << std::endl;
	debug_device_ptr(onGPU->X, onGPU->numVertices);
#endif
	//use weights stored in X to
	//compute additional cost of this step of min spanning tree.
	//note: moveWeights set to 0 elements S[S[i]] = i;

	thrust::device_ptr<unsigned int> Ws(onGPU->X);
	thrust::inclusive_scan(Ws, Ws + onGPU->numVertices, Ws);

	hipDeviceSynchronize(); 


#ifdef PEDANTIC
	std::cout << "sum of min weights outgoing edges performed on X (w,v)" << std::endl;
	debug_device_ptr(onGPU->X, onGPU->numVertices);
#endif
	//last cell of F contains the cost.
	//Load into a variable.
	unsigned int deltaCosts = 0;
	hipMemcpy(&deltaCosts, onGPU->X + onGPU->numVertices - 1, sizeof(unsigned int), hipMemcpyDeviceToHost);
#if defined(DEBUG ) || defined(PEDANTIC)
	std::cout << "Additional costs" << deltaCosts << std::endl;
#endif
	//update the current MST cost.
	onGPU->cost += deltaCosts;
}

void saveMinOutgoingEdges(DatastructuresOnGpu* onGPU) {

#ifdef PEDANTIC
	std::cout << "Min edge ids:" << std::endl;
	debug_device_ptr(onGPU->F, onGPU->numVertices);
#endif

	hipMemcpy(onGPU->edgeIDresult + onGPU->savedEdges, onGPU->F, sizeof(unsigned int)*onGPU->numVertices, hipMemcpyDeviceToDevice);
	onGPU->savedEdges += onGPU->numVertices;
}

void buildSuccessor(DatastructuresOnGpu* onGPU) {
#ifdef PEDANTIC
	std::cout << "Successor: " << std::endl;
	debug_device_ptr(onGPU->S, onGPU->numVertices);
#endif
	//compute successor

	if (sizeof(unsigned int) * onGPU->numVertices < onGPU->maxSharedBytes) {
		int block_dim = std::min(1024, blockRounder((int)(onGPU->numVertices + 16) / 17));
		replaceTillFixedPointInShared << <1, block_dim, sizeof(unsigned int) * onGPU->numVertices >> >(onGPU->S, onGPU->numVertices);
	}
	else {
		unsigned int max_vertex_in_shared = onGPU->maxSharedBytes / sizeof(unsigned int);
		replaceTillFixedPoint <<<1, 1024,  max_vertex_in_shared* sizeof(unsigned int) >>>(onGPU->S, onGPU->numVertices, max_vertex_in_shared);
	}
	hipDeviceSynchronize(); 

#ifdef PEDANTIC
	std::cout << "Successor after fixed point: " << std::endl;
	debug_device_ptr(onGPU->S, onGPU->numVertices);
#endif
}

void buildSupervertexId(DatastructuresOnGpu* onGPU) {
	thrust::device_ptr<unsigned int>
		s_on_gpu(onGPU->S), v_on_gpu(onGPU->vertices),
		f_on_gpu(onGPU->F);
	//1. aggregate vertices with the same supervertex id
	//   in order to be able to reconstruct which supervertex is related to which vertex
	//	 we move according to the transformation the original position in which is was. 
	//	 The original position will be temporarely stored in v.
#ifdef PEDANTIC
	std::cout << "Rebuild graph representation for the next iteration" << std::endl;
	std::cout << "Source: " << std::endl;
	debug_device_ptr(onGPU->vertices, onGPU->numVertices);
	std::cout << "Renaming to supervertex: " << std::endl;
	debug_device_ptr(onGPU->S, onGPU->numVertices);
#endif
	fill << < grid(onGPU->numVertices, BLOCK_SIZE), BLOCK_SIZE >> >(onGPU->vertices, 1, onGPU->numVertices);
	hipDeviceSynchronize(); 
	//use exclusive scan so that first element get 0 as original position
	thrust::exclusive_scan(v_on_gpu, v_on_gpu + onGPU->numVertices, v_on_gpu);
	hipDeviceSynchronize();

	//2. order according to successor and move accordingly the array v
	thrust::sort_by_key(s_on_gpu, s_on_gpu + onGPU->numVertices, v_on_gpu);
	hipDeviceSynchronize();

	//3. create a flag that contains 1 in position i , when  supervertex[i-1] != supervertex[i] , 0 otherwise.
	hipMemset(onGPU->F, 0, onGPU->numVertices * sizeof(unsigned int));
	hipDeviceSynchronize(); 
	mark_discontinuance << < grid(onGPU->numVertices, BLOCK_SIZE), BLOCK_SIZE >> >(onGPU->F, onGPU->S, onGPU->numVertices);
	hipDeviceSynchronize(); 

#ifdef PEDANTIC
	std::cout << "Pick different supervertex" << std::endl;
	std::cout << "vertex index: " << std::endl;
	debug_device_ptr(onGPU->vertices, onGPU->numVertices);
	std::cout << "supervertex: " << std::endl;
	debug_device_ptr(onGPU->S, onGPU->numVertices);
	std::cout << "mark supervetex end: " << std::endl;
	debug_device_ptr(onGPU->F, onGPU->numVertices);
#endif

	//4. perform a scan to build supervertex identifier.
	thrust::inclusive_scan(f_on_gpu, f_on_gpu + onGPU->numVertices, f_on_gpu);
	hipDeviceSynchronize(); 

#ifdef PEDANTIC
	std::cout << "id of supervetex: " << std::endl;
	debug_device_ptr(onGPU->F, onGPU->numVertices);
#endif
	//5. once that for each vertex we have determined the new supervetex id.
	//	 Reconstruct a map that permits to move from the vertex->new_supervertex_id. 
	//   The map will be placed in S. 
	//use v that have been reordered to recreate a map v->new_supervertex_id according to the original order.

	thrust::scatter(f_on_gpu, f_on_gpu + onGPU->numVertices, v_on_gpu, s_on_gpu);
	hipDeviceSynchronize(); 
#ifdef PEDANTIC
	std::cout << "vertex ->id of supervetex: " << std::endl;
	debug_device_ptr(onGPU->S, onGPU->numVertices);
#endif
}



void orderUVW(DatastructuresOnGpu* onGPU) {
	thrust::device_ptr<unsigned int>
		s_on_gpu(onGPU->S), v_on_gpu(onGPU->vertices),
		f_on_gpu(onGPU->F), e_on_gpu(onGPU->edges),
		w_on_gpu(onGPU->weights), eptr_on_gpu(onGPU->edgePtr),
		x_on_gpu(onGPU->X), edgeId_on_gpu(onGPU->edgeID);
	thrust::device_ptr<NVEcell> NVE_on_gpu(onGPU->NVE);

	//1. consider the arrays that describe the edges u-w->v.
	//1.a remap u,v with new supervertex id.
	//	v = source
	//	edges = destination
	//	weights = weights
	hipMemset(onGPU->F, 0, onGPU->numEdges* sizeof(unsigned  int));
	//since elements 0 has to obtain flag = 0 use edgePTR excluding the first cell in the mark_edge_ptr call
	mark_edge_ptr << <grid(onGPU->numVertices, BLOCK_SIZE), BLOCK_SIZE >> > (onGPU->F, onGPU->edgePtr, onGPU->numVertices);
	hipDeviceSynchronize(); 
#ifdef PEDANTIC
	std::cout << "=============================== reorder according to U,V,W where U-w->V : ===============" << std::endl;
	std::cout << "Edge ptr: " << std::endl;
	debug_device_ptr(onGPU->edgePtr, onGPU->numVertices);
	std::cout << "set 1 at start of the edge segments to build source vertex: " << std::endl;
	debug_device_ptr(onGPU->F, onGPU->numEdges);
#endif
	//after inclusive scan f[i] = index of source vertex

	thrust::inclusive_scan(f_on_gpu, f_on_gpu + onGPU->numEdges, f_on_gpu);
	hipDeviceSynchronize(); 
#ifdef PEDANTIC
	std::cout << "source of the edge:" << std::endl;
	debug_device_ptr(onGPU->F, onGPU->numEdges);
	std::cout << "destination of the edge:" << std::endl;
	debug_device_ptr(onGPU->edges, onGPU->numEdges);
	std::cout << "Map:" << std::endl;
	debug_device_ptr(onGPU->S, onGPU->numVertices);
#endif
	//convert edge source & destination thanks to vector S.
	thrust::gather(f_on_gpu, f_on_gpu + onGPU->numEdges, s_on_gpu, v_on_gpu);
	thrust::gather(e_on_gpu, e_on_gpu + onGPU->numEdges, s_on_gpu, x_on_gpu);
	//we would like to store all info in e.
	//mem cpy is necessary due to the fact that gather input /output has to reside on different memory areas
	hipMemcpy(onGPU->edges, onGPU->X, sizeof(unsigned int)*onGPU->numEdges, hipMemcpyDeviceToDevice);

#ifdef PEDANTIC
	std::cout << "Transformated source of the edge (Result)  " << std::endl;
	debug_device_ptr(onGPU->vertices, onGPU->numEdges);
	std::cout << "Transformated destination of the edge (Result)" << std::endl;
	debug_device_ptr(onGPU->edges, onGPU->numEdges);
#endif
	//2. reored triplet u,v,w. But instead of using a single array . 
	//	 Exploit a stable sort and a perform three different order carrying along a index represented by X.
	//   as in radix sort go from least signicants bits to most significative bits,
	//2.a. create index increment.
	fill <<< grid(onGPU->numEdges, BLOCK_SIZE), BLOCK_SIZE >>>(onGPU->X, 1, onGPU->numEdges);
	hipDeviceSynchronize(); 
	//2.b. create index by accumulating increments: use exclusive since first element has index 0.
	thrust::exclusive_scan(x_on_gpu, x_on_gpu + onGPU->numEdges, x_on_gpu);
	hipDeviceSynchronize(); 

	hipMemset(onGPU->NVE, 0, onGPU->numEdges* sizeof(NVEcell));
	//int block_dim = std::min(1024, blockRounder((int)(onGPU->numVertices + 16) / 17));
	loadNVEsingle <<<  grid(onGPU->numEdges, BLOCK_SIZE), BLOCK_SIZE >>>(onGPU->NVE, onGPU->vertices, onGPU->edges, onGPU->weights, onGPU->numEdges);


#ifdef PEDANTIC
	std::cout << "Keys(X) before ordering" << std::endl;
	debug_device_ptr(onGPU->X, onGPU->numEdges);
	std::cout << "Source vertices" << std::endl;
	debug_device_ptr(onGPU->vertices, onGPU->numEdges);
	std::cout << "Destination vertices" << std::endl;
	debug_device_ptr(onGPU->edges, onGPU->numEdges);
	std::cout << "Weights" << std::endl;
	debug_device_ptr(onGPU->weights, onGPU->numEdges);
	std::cout << "Edges ids:" << std::endl;
	debug_device_ptr(onGPU->edgeID, onGPU->numEdges);
#endif

	thrust::sort_by_key(NVE_on_gpu, NVE_on_gpu + onGPU->numEdges, x_on_gpu);

	//3.e. last: in order for i that:	f[i] < f[j] and 
	//								edges[i] < edges[j] and
	//								weights[i] < weights[j] 
	// we have to reorder edges and weights arrays according to index x.
	//NOTE: nothing has been said about gather values and output possible overlapping.
	unloadNVE <<<  grid(onGPU->numEdges, BLOCK_SIZE), BLOCK_SIZE >> >(onGPU->NVE, onGPU->vertices, onGPU->edges, onGPU->weights, onGPU->numEdges);
	thrust::gather(x_on_gpu, x_on_gpu + onGPU->numEdges, edgeId_on_gpu, s_on_gpu);
	hipMemcpy(onGPU->edgeID, onGPU->S, onGPU->numEdges* sizeof(unsigned int), hipMemcpyDeviceToDevice);


#ifdef PEDANTIC
	std::cout << "Keys(X) after ordering source, destination, weights" << std::endl;
	debug_device_ptr(onGPU->X, onGPU->numEdges);
	std::cout << "Source vertices" << std::endl;
	debug_device_ptr(onGPU->vertices, onGPU->numEdges);
	std::cout << "Destination vertices" << std::endl;
	debug_device_ptr(onGPU->edges, onGPU->numEdges);
	std::cout << "Weights" << std::endl;
	debug_device_ptr(onGPU->weights, onGPU->numEdges);
	std::cout << "Edges ids:" << std::endl;
	debug_device_ptr(onGPU->edgeID, onGPU->numEdges);
#endif

}

void rebuildEdgeWeights(DatastructuresOnGpu* onGPU) {
	thrust::device_ptr<unsigned int>
		s_on_gpu(onGPU->S), v_on_gpu(onGPU->vertices),
		f_on_gpu(onGPU->F), e_on_gpu(onGPU->edges),
		w_on_gpu(onGPU->weights), eptr_on_gpu(onGPU->edgePtr),
		x_on_gpu(onGPU->X), edgeID_on_gpu(onGPU->edgeID);


	//8. create Edge, weights
	//8.a use F to mark subseqent (v,u) which are not equals and neither self-loops.
	hipMemset(onGPU->F, 0, onGPU->numEdges* sizeof(unsigned int));
	hipDeviceSynchronize(); 
	mark_differentUV << <grid(onGPU->numEdges, BLOCK_SIZE), BLOCK_SIZE >> >(onGPU->F, onGPU->vertices, onGPU->edges, onGPU->numEdges);
	hipDeviceSynchronize(); 
	//8.b perform a scan of F to obtain position were to put edge[idx], weight[idx] put the scan result on X
	//	  then use: 
	//		-	x as a map 
	//		-   F as a stencil
	thrust::inclusive_scan(f_on_gpu, f_on_gpu + onGPU->numEdges, x_on_gpu);
	//8.c compute the number of edges available in the next iteration by reading the tail of the scan vector result.

	hipMemcpy(&onGPU->newNumEdges, onGPU->X + onGPU->numEdges - 1, sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(&onGPU->newNumVertices, onGPU->vertices + onGPU->numEdges - 1, sizeof(unsigned int), hipMemcpyDeviceToHost);
	onGPU->newNumVertices++;

	//8.d. since index start at 0 subtract 1 from all the indices.
	thrust::transform(x_on_gpu, x_on_gpu + onGPU->numEdges, x_on_gpu, minus1());
	hipDeviceSynchronize(); 
#ifdef PEDANTIC
	std::cout << "new edges/weights" << std::endl;
	std::cout << "mark relevant edges for edges/weights" << std::endl;
	debug_device_ptr(onGPU->F, onGPU->numEdges);
	std::cout << "positions" << std::endl;
	debug_device_ptr(onGPU->X, onGPU->numEdges);
#endif
	//Since there are multiple elements with the same destination index and we have to insert only those that are in correspondence with a change in x.
	//we exploit a version of scatter that expect a stencil.
	//8.d. move weights
	//since input data and output can't reside on the same memory area we move them temporarily to S
	hipMemcpy(onGPU->S, onGPU->weights, sizeof(unsigned int)* onGPU->numEdges, hipMemcpyDeviceToDevice);
	thrust::scatter_if(s_on_gpu, s_on_gpu + onGPU->numEdges, x_on_gpu, f_on_gpu, w_on_gpu);
	//8.e. move destination vertices
	//since input data and output can't reside on the same memory area we mve them temporarily to S
	hipMemcpy(onGPU->S, onGPU->edges, sizeof(unsigned int)* onGPU->numEdges, hipMemcpyDeviceToDevice);
	thrust::scatter_if(s_on_gpu, s_on_gpu + onGPU->numEdges, x_on_gpu, f_on_gpu, e_on_gpu);
	hipDeviceSynchronize(); 
	//8.f. move edge IDs
	//since input data and output can't reside on the same memory area we mve them temporarily to S
	hipMemcpy(onGPU->S, onGPU->edgeID, sizeof(unsigned int)* onGPU->numEdges, hipMemcpyDeviceToDevice);
	thrust::scatter_if(s_on_gpu, s_on_gpu + onGPU->numEdges, x_on_gpu, f_on_gpu, edgeID_on_gpu);
	hipDeviceSynchronize();
	//8.g. move source
	//since input data and output can't reside on the same memory area we mve them temporarily to S
	hipMemcpy(onGPU->S, onGPU->vertices, sizeof(unsigned int)* onGPU->numEdges, hipMemcpyDeviceToDevice);
	thrust::scatter_if(s_on_gpu, s_on_gpu + onGPU->numEdges, x_on_gpu, f_on_gpu, v_on_gpu);

#ifdef PEDANTIC
	std::cout << "new edges (" << onGPU->newNumEdges << ") :" << std::endl;
	std::cout << "new sources" << std::endl;
	debug_device_ptr(onGPU->vertices, onGPU->newNumEdges);
	std::cout << "new weights" << std::endl;
	debug_device_ptr(onGPU->weights, onGPU->newNumEdges);
	std::cout << "new edges" << std::endl;
	debug_device_ptr(onGPU->edges, onGPU->newNumEdges);
	std::cout << "New edge id" << std::endl;
	debug_device_ptr(onGPU->edgeID, onGPU->newNumEdges);
#endif
}

void rebuildEdgePtr(DatastructuresOnGpu* onGPU) {
	thrust::device_ptr<unsigned int>
		s_on_gpu(onGPU->S), v_on_gpu(onGPU->vertices),
		f_on_gpu(onGPU->F),
		w_on_gpu(onGPU->weights), eptr_on_gpu(onGPU->edgePtr),
		x_on_gpu(onGPU->X);

	if (onGPU->newNumEdges == 0)
		return;

	//exploit data already computed for Edges in X number of useful edges
	//9. create EdgePTR:
	//9.a find discontinuance in source vertex
	//    since there was a collapsing in the previous step there would always be a edge for each supervertex.
	hipMemset(onGPU->F, 0, onGPU->newNumEdges* sizeof(unsigned int));
	hipDeviceSynchronize(); 
	mark_differentU << <grid(onGPU->newNumEdges, BLOCK_SIZE), BLOCK_SIZE >> >(onGPU->F, onGPU->vertices, onGPU->newNumEdges);
	hipDeviceSynchronize(); 


	//9.d Build edge ptr:
	//when the source vector change pick the number of relevant edges till the previous vertex and then increment by one 
	fill<<< grid(onGPU->newNumEdges, BLOCK_SIZE), BLOCK_SIZE>>>(onGPU->X, 1, onGPU->newNumEdges);
	thrust::exclusive_scan(x_on_gpu, x_on_gpu + onGPU->newNumEdges, x_on_gpu);

	//if on x we have the counted the number of edges that will be preserved
	//   on f we have the stencil that indicate two different consequence sources
	//   on v we have the scan result on the stencil(i.e. v itself).
#ifdef PEDANTIC
	std::cout << "mark relevant edges for vertices" << std::endl;
	debug_device_ptr(onGPU->F, onGPU->numEdges);
	std::cout << "positions" << std::endl;
	debug_device_ptr(onGPU->S, onGPU->numEdges);
	std::cout << "scan result on relevant edges (already performed at step 7)" << std::endl;
	debug_device_ptr(onGPU->X, onGPU->numEdges);
#endif

	thrust::scatter_if(x_on_gpu, x_on_gpu + onGPU->newNumEdges, v_on_gpu, f_on_gpu, eptr_on_gpu);
	hipDeviceSynchronize(); 
	
#ifdef PEDANTIC
	std::cout << "new edge ptr" << std::endl;
	debug_device_ptr(onGPU->edgePtr, onGPU->newNumVertices);
#endif



}


